#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <cstdio>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
using namespace std;

void print_traj(FILE* out_traj,double* traj,int N_spots);
__global__ void perform_sweeps(double* d_traj, int N_spots, double a, double omega,
	double bot,double x0, double sigma_coef, double sigma_sweeps_period,
	double acc_rate_up_border, double acc_rate_low_border, int N_sweeps_waiting, hiprandState *rng_states);

int main()
{
    clock_t start,end;
	start=clock();

	const int N_sweeps_waiting=800000;
	const double a=0.08;
	const int N_spots=1024;
	//double beta=a*N_spots;
	const double omega=7.0;
	double bot=1.0;
	double x0=bot;

	const int sigma_local_updates_period=2000;
	const int sigma_sweeps_period=ceil((double)sigma_local_updates_period/N_spots);
	const double sigma_coef=1.2;
	const double acc_rate_up_border=0.3;
	const double acc_rate_low_border=0.2;

	FILE *out_traj;
	out_traj=fopen("out_traj.txt","w");

	double* h_traj;
	h_traj=(double*)malloc(N_spots*sizeof(double));
	double* d_traj;
	hipMalloc((void**)&d_traj, (N_spots*sizeof(double));

	dim3 grid(1,1,1);
	dim3 block(N_spots,1,1);
	
	hiprandState *devStates;
    hipMalloc((void**)&devStates, N_spots*sizeof(hiprandState));

	perform_sweeps(d_traj, N_spots, a, omega, bot, x0, sigma_coef, sigma_sweeps_period,
		acc_rate_up_border, acc_rate_low_border, N_sweeps_waiting, devStates);
	cudamMemcpy(h_traj,d_traj,N_spots*sizeof(double));

	print_traj(out_traj,h_traj,N_spots);

	fclose(out_traj);
    end=clock();
	printf("TIME: %.2lf ms\n",(double)(end-start)/CLOCKS_PER_SEC*1000);
}
